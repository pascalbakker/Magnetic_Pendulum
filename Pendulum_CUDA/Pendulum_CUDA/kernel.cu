#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "stdafx.h"
#include <math.h>
#include <string.h>
#include <assert.h>
#include <stdlib.h>
#include <memory.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

//Holds RGB values
struct rgb {
	int r, g, b;
};
#define ORIGINDIST 5.0 //distance the magnets are from x: 0 y:0
#define NUM_MAGNETS 3
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
//Code constants
const int iterations = 5000; //number of iterations for integration
const int minSteps = 300; //Minimum number of steps before pendulum() breaks

						  //Image Parameters
const unsigned imageWidth = 300;
const unsigned imageHeight = 300;

inline void gpuAssert(hipError_t code, char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
	}
}

// Determines which magnet the pendulumn is going to land
__device__ int pendulum(double x, double y, double *dev_magnets) {
	const float MAXFLOAT = 99999999.9;
	int closest_magnet = -1, ct;
	double *tmp, *acc_p, *acc, *acc_n, t, dt, closest_dist, dist;
	const double k_f = 0.001; //friction constant
	const double k_g = 0.0;  //pendulum constant
	const double k_m = 0.3; //magnet force constant
	const double m_fHeight = 1.0; //pendulum height above magnets
								  //All force sources positions
								  // value 1: magnet's x coordinate
								  // value 2: magnet's y coordinate
								  //Coordinate System parameters

								  //Starting Vectors
	double pos[2] = { x,y };
	double vel[2] = { 0,0 }; //starting velocity
	double r[2] = { 0,0 }; //starting position
	double acc0[2] = { 0,0 }; //pendulum acceleration
	double acc1[2] = { 0,0 }; //pendulumn accelertion next step
	double acc2[2] = { 0,0 }; //accel prev step
	double force[2] = { 0,0 }; //x & y forces on pendulum
	double *src;

	//tmp = malloc(2 * sizeof(double));

	acc_n = acc2;

	acc_p = acc0;

	acc = acc1;

	//Time variables
	t = 0;
	dt = 0.1;

	//Calculate source's forces
	for (ct = 0; ct < iterations; ++ct) {
		//Update time
		t += dt;
		//Update position
		pos[0] += vel[0] * dt + dt * dt * (2.0 / 3.0 * acc[0] - 1.0 / 6.0 * acc_p[0]);
		pos[1] += vel[1] * dt + dt * dt * (2.0 / 3.0 * acc[1] - 1.0 / 6.0 * acc_p[1]);

		//reset acceleration
		acc_n[0] = 0.0;
		acc_n[1] = 0.0;

		closest_dist = MAXFLOAT;
		//Calculate the magnet and pendulum forces on the pendulumn acceleration
		for (int i = 0; i < NUM_MAGNETS + 1; i++) {
			src = dev_magnets;

			r[0] = pos[0] - src[0];
			r[1] = pos[1] - src[1];

			//Calculate force
			//Force of mountpoint on pendulum
			if (i == 0) {
				force[0] = k_g * r[0];
				force[1] = k_g * r[1];
				//magnetic force
			}
			else {
				dist = sqrt((src[0] - pos[0])*(src[0] - pos[0]) + (src[1] - pos[1])*(src[1] - pos[1]) + m_fHeight * m_fHeight);

				if (dist < closest_dist) {
					closest_dist = dist;
					closest_magnet = i;
				}
				force[0] = k_m / (dist*dist*dist)*r[0];
				force[1] = k_m / (dist*dist*dist)*r[1];
			}

			//Update acceleration
			acc_n[0] -= force[0];
			acc_n[1] -= force[1];

			//Break case
			if (ct < minSteps&&fabs(r[0]) < 0.1&&fabs(r[1]) < 0.1&&fabs(vel[0]) < 0.05&&fabs(vel[1]) < 0.05) {
				break;
			}
		}

		// Friction force on pendulum
		acc_n[0] -= vel[0] * k_f;
		acc_n[1] -= vel[1] * k_f;

		// Update velocity
		vel[0] += dt * (1.0 / 3.0 * acc_n[0] + 5.0 / 6.0 * acc[0] - 1.0 / 6.0 * acc_p[0]);
		vel[1] += dt * (1.0 / 3.0 * acc_n[1] + 5.0 / 6.0 * acc[1] - 1.0 / 6.0 * acc_p[1]);

		//Store values
		tmp = acc_p;
		acc_p = acc;
		acc = acc_n;
		acc_n = tmp;
	}
	return closest_magnet;
}

// Creates a square box around the magnet
__device__ int isNearMagnet(double x, double y, double *dev_magnets) {
	double r[2], *magnet;
	int i;
	for (i = 1; i < NUM_MAGNETS + 1; i++) {
		magnet = dev_magnets;
		r[0] = magnet[0] - x;
		r[1] = magnet[1] - y;
		if (fabs(r[0]) < 0.4&&fabs(r[1]) < 0.4) {
			return 1;
		}
	}
	return 0;
}


// Creates image with pixel values calculated from pendulum function and prints out color distribution
void printImage(char* imageName, struct rgb *mat) {
	char *location, *exten;
	//CREATE IMAGE WITH RGB MATRIX
	location = "C:/Users/rcall/source/repos/CUDA_Final/ConsoleApplication1/gif1/";
	exten = ".png";

	char *name = (char*)malloc(1 + strlen(location) + strlen(imageName) + strlen(exten));

	if (name != 0) {
		strcpy(name, location);
		strncat(name, imageName, 1);
		strncat(name, exten, 4);
	}
	else {
		printf("Variable name is Null exiting.");
		exit(0);
	}
	FILE * fp = fopen(name, "wb");
	if (fp == NULL) {
		printf("File %s does not exist.", name);
		exit(0);
	}
	else {
		fprintf(fp, "P3\n");
		fprintf(fp, "%d %d\n", imageWidth, imageHeight);
		fprintf(fp, "%d\n", 255);
	}
	for (unsigned i = 0; i < imageHeight; i++) {
		for (unsigned j = 0; j < imageWidth; j++) {
			fprintf(fp, "%d ", mat[(j + i * imageWidth)].r);
			fprintf(fp, "%d ", mat[(j + i * imageWidth)].g);
			fprintf(fp, "%d    ", mat[(j + i * imageWidth)].b);

			if ((i + j) % 13 == 0) fprintf(fp, "\n");
		}
	}
	fclose(fp);
}

__global__ void Calculate(struct rgb *dev_mat, double *dev_magnets) {
	const double minX = -15.0;
	const double maxX = 15.0;
	const double minY = -15.0;
	const double maxY = 15.0;

	//CUDA Constraints
	unsigned r = threadIdx.x + (blockDim.x*blockIdx.x);
	unsigned i = threadIdx.y + (blockDim.y*blockIdx.y);

	//Image matrix
	int result;
	double cx, cy;

	//Referenced Schuster's Mandlebrot slides
	if ((r < imageHeight) && (i < imageWidth)) {
		cx = minX + (r*1.0 / imageHeight)*(maxX - minX); //constant real
		cy = minY + (i * 1.0 / imageWidth) * (maxY - minY);
		//Calculate coordinate result
		result = pendulum(cx, cy, dev_magnets);
		//determine color
		struct rgb color;
		color.r = 0;
		color.g = 0;
		color.b = 0;

		if (isNearMagnet(cx, cy, dev_magnets)) {//If coordinate is magnet
			color.r = 255;
			color.b = 255;
		}
		else {
			switch (result) {
			case 1: color.r = 255;
				break;
			case 2: color.g = 255;
				break;
			case 3: color.b = 255;
				break;
			case 4:
				color.r = 255;
				color.g = 255;
				break;
			case 5:
				color.g = 255;
				color.b = 255;
				break;
			case 6: color.r = 127;
				break;
			case 7: color.g = 127;
				break;
			case 8: color.b = 127;
				break;
			}
		}
		//add color to matrix
		dev_mat[(i + r * imageWidth)] = color;
	}
	//for (int i = 0; i < 3; i++) {
	//	for (int j = 0; j < 3; j++) {
	//		printf("%d ", dev_magnets[i][j]);
	//	}
	//	printf("\n");
	//}
}


//Execution method
int main() {
	printf("Begin\n");
	int numPics = 100;
	double yval = (2 * (ORIGINDIST + 15)) / numPics;
	size_t bytes = imageHeight * imageWidth * sizeof(struct rgb)*sizeof(size_t);
	size_t mat_size = sizeof(double) * size_t(NUM_MAGNETS + 1 * 2);

	struct rgb *dev_mat;
	double  *dev_magnets;
	struct rgb *h_mat = (struct rgb*)malloc(bytes);
	double h_magnets[NUM_MAGNETS + 1][2] = { { 0,0 },{ 0,-ORIGINDIST - 15 },{ -ORIGINDIST - 0.5,0 },{ ORIGINDIST + 0.5,0 } };

	struct rgb *mat = (struct rgb*)malloc(bytes);
	double magnets[NUM_MAGNETS + 1][2] = { { 0,0 },{ 0,-ORIGINDIST - 15 },{ -ORIGINDIST - 0.5,0 },{ ORIGINDIST + 0.5,0 } };

	gpuErrchk(hipMalloc((void**)&dev_mat, bytes));
	gpuErrchk(hipMalloc((void**)&dev_magnets, bytes));
	gpuErrchk(hipMemcpy(dev_mat, mat, bytes, hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(dev_magnets, magnets, mat_size, hipMemcpyHostToDevice));

	for (int i = 0; i < numPics + 1; i++) {
		printf("Running image %d.png\n", i);
		char array[12];
		sprintf(array, "%d", i);
		printf("Image: %s.pgm\n", array);
		magnets[1][1] = magnets[1][1] + yval;
		Calculate << <40, 20 >> > (dev_mat, dev_magnets);
		hipDeviceSynchronize();
		gpuErrchk(hipMemcpy(h_mat,dev_mat, bytes, hipMemcpyDeviceToHost));
		gpuErrchk(hipMemcpy(h_magnets, dev_magnets, mat_size, hipMemcpyDeviceToHost));
		printf("Test Magnet %i: %f\n", i, h_magnets);
		printImage(array, mat);
	}
	printf("Done");
	return 0;
}